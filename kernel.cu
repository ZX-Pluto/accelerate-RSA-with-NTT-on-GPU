#include "hip/hip_runtime.h"
﻿#include "common.c"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_fp16.h"
#include <iostream>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>

using namespace std;
typedef long long ll;

const int  P = 23068673, G = 3;
int ebits, dbits, nbits;

//input:a k 
//output:a^k mod P
inline ll power(ll a, ll k) {
	ll base = 1;
	for (; k; k >>= 1) {
		if (k & 1) base = (base * a) % P;
		a = (a * a) % P;
	}
	return base;
}

//input:bignum bn 
//output:bits of bn
int bignum_numbits(uint32_t* bn) {

	register int i = (64 << 5) - 1;
	for (; i > 0; --i)
	{
		if ((bn[i >> 5] >> (i & 0b11111)) & 1)
			return i + 1;
	}
	return 0;
}

//read bignum from string
void bignum_from_string(uint32_t* bn, char* str, int nhex)
{
	memset(bn, 0, nhex / 2);

	uint32_t tmp;                        
	int i = nhex - 8;				/* index into string */
	int j = 0;						/* index into array */

	/* reading last hex-byte "MSB" from string first -> big endian */
	/* MSB ~= most significant byte / block ? :) */
	while (i >= 0)
	{
		tmp = 0;
		sscanf(&str[i], "%8x", &tmp);//read one word
		bn[j] = tmp;
		i -= 8; /* step WORD_SIZE hex-byte(s) back in the string. */
		j += 1; /* step one element forward in the array. */
	}
}

__global__ void PrintOnGpu(uint32_t* X)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	printf("[%d]:%u ", tid, X[tid]);
}

//input:a b 
//output:a + b 
__global__ void AddOnGPU(uint32_t* Device_A, uint32_t* Device_B)
{
	__shared__ int flag;
	flag = 0;
	extern __shared__ int carry[];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t temp;
	carry[tid] = 0;
	temp = Device_A[tid] + Device_B[tid];
	carry[tid + 1] = temp >> 32;
	if (carry[tid])
		flag = 1;
	Device_A[tid] = temp;
	for (; flag == 1; )
	{
		flag = 0;
		temp = Device_A[tid] + carry[tid];
		carry[tid + 1] = temp >> 32;
		if (carry[tid])
			flag = 1;
		Device_A[tid] = temp;
	}
}

//input:a b 
//output:a - b 
__global__ void SubOnGPU(uint32_t* Device_A, uint32_t* Device_B, int* cmp)
{
	if (*cmp != -1)
	{		
		extern __shared__ uint32_t borrow[];
		int tid = blockIdx.x * blockDim.x + threadIdx.x;
		uint64_t res;
		borrow[tid] = 0;

		res = (uint64_t)Device_A[tid] + 0x100000000 - (uint64_t)Device_B[tid];
		borrow[tid + 1] = (res >> 32) ^ 1;
		Device_A[tid] = res;
		__syncthreads();
		for (; borrow[tid] == 1; )
		{		
			res = (uint64_t)Device_A[tid] + 0x100000000 - (uint64_t)borrow[tid];
			borrow[tid] = 0;
			borrow[tid + 1] = (res >> 32) ^ 1;
			Device_A[tid] = res;
		}
		__syncthreads();
	}
}

//input:a b flag 
//output: flag = (a > b) ? 1:0:-1
__global__ void CmpOnGPU(uint32_t* Device_A, uint32_t* Device_B,int size,int* flag)
{
	*flag = 0;
	for (int i = size; i >= 0; i--)
	{
		if (Device_A[i] > Device_B[i])
		{
			*flag = 1;
			break;
		}
		else if (Device_A[i] < Device_B[i])
		{
			*flag = -1;
			break;
		}
	}
}

//ntt
__global__ void NttOnGPU(uint32_t* Device_A, ll* g, int limit) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t x, y;
	int j, m;
	for (m = 1; m < limit && m < 32; m <<= 1) {
		j = tid / m % 2;
		x = (ll)Device_A[tid] * g[j * limit / (2 * m) * (tid % m)] % P;
		Device_A[tid] = ((1 - 2 * j) * x + __shfl_xor(x, m) + j * P) % P;
	}
	for (; m < limit; m <<= 1) {
		j = tid / m % 2;
		if (j)
		{
			x = (ll)Device_A[tid] * g[limit / (2 * m) * (tid % m)] % P;
			Device_A[tid] = x;
		}
		else
		{
			x = Device_A[tid];
		}
		__syncthreads();
		y = Device_A[tid ^ m];
		Device_A[tid] = ((1 - 2 * j) * x + y + j * P) % P;
	}
}

//inverse ntt
__global__ void _NttOnGPU(uint32_t* Device_A, ll* g, int limit) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t x, y;
	int j, m;
	for (m = 1; m < limit && m < 32; m <<= 1) {
		j = tid / m % 2;
		x = (ll)Device_A[tid] * g[j * ((limit - (limit / (2 * m) * (tid % m))) % limit)] % P;
		Device_A[tid] = ((1 - 2 * j) * x + __shfl_xor(x, m) + j * P) % P;
	}

	for (; m < limit; m <<= 1) {
		j = tid / m % 2;
		if (j)
		{
			x = (ll)Device_A[tid] * g[(limit - (limit / (2 * m) * (tid % m))) % limit] % P;
			Device_A[tid] = x;
		}
		else
		{
			x = Device_A[tid];
		}
		__syncthreads();
		y = Device_A[tid ^ m];
		Device_A[tid] = ((1 - 2 * j) * x + y + j * P) % P;
	}
}

//input:a b 
//output:a * b mod P
__global__ void MulOnGPU(uint32_t* Device_A, uint32_t* Device_B) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	Device_A[tid] = ((ll)Device_A[tid] * (ll)Device_B[tid]) % P;
}

//input:ntt(a) ntt(b) 
//output:ntt(a) + ntt(b)
__global__ void NttAddOnGPU(uint32_t* Device_A, uint32_t* Device_B) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	Device_A[tid] = Device_A[tid] + Device_B[tid];
}

//butterfly in gpu
__global__ void ButOnGPU(uint32_t* Device_A, int* D_rev) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	Device_A[tid] = Device_A[D_rev[tid]];
}

//input:a inv output:a/n mod p
__global__ void MulDivOnGPU(uint32_t* Device_A, ll inv) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	Device_A[tid] = (ll)Device_A[tid] * inv % P;
}

//bignum to polynomial
__global__ void BignumToPolynomial(uint8_t* x, uint32_t* X, int limit) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	X[tid] = x[tid];
	X[tid + limit] = 0;
}

//polynomial to bignum and mod R
__global__ void PolynomialToBignum(uint8_t* x, uint32_t* X, int offset, int nbytes, int nbits) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (nbits != 0)
	{
		int z = 8 - nbits;
		x[tid] = ((uint8_t)X[tid + nbytes]) >> nbits;
		x[tid] += ((uint8_t)X[tid + nbytes + 1]) << z;
	}
	else
	{
		x[tid] = X[tid + nbytes];
	}
	x[tid + offset] = 0;

}

//polynomial mod R
__global__ void PolynominalModR(uint32_t* X, uint32_t* Out, int limit,int nbits) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	Out[tid] = X[tid];
	Out[tid + limit] = 0;
	Out[nbits >> 3] = Out[nbits >> 3] % (1 << (nbits & 0b111));
	//Out[limit - 1] = Out[limit - 1] & 0b1111;
}

//calculate carry 
__global__ void PolynominalCarry(uint32_t* X, int limit)
{
	for (int i = 0; i < limit; i++)
	{
		X[i + 1] += X[i] >> 8;
		X[i] = X[i] & 0xFF;
	}
}

//global val

uint32_t* nn, * e, * d, * r2m, * _n, * val;
uint8_t* x, * y;
uint32_t* n_ntt, * _n_ntt, * M;
ll inv;
int* d_rev;
ll* d_root;
ll root[1 << 11] = { 0 };
int rev[1 << 12] = { 0 };

hipStream_t stream[16];
uint32_t* XX[16], * YY[16];
uint8_t* xx[16], * yy[16];
uint32_t* Out[16];
int* flag[16];

// montgomery mul with ntt
void NTTMonMulOnGPU(hipStream_t stream, uint8_t* x, uint8_t* y, uint32_t* out, uint32_t* X, uint32_t* Y, int limit, int* flag)
{
	BignumToPolynomial << <1, limit / 2, 0, stream >> > (x, X, limit / 2);
	BignumToPolynomial << <1, limit / 2, 0, stream >> > (y, Y, limit / 2);

	/*
	if (test)
	{
		PrintOnGpu << <1, limit, 0, stream >> > (X);
		hipStreamSynchronize(stream);
		printf("\n\n");
		PrintOnGpu << <1, limit, 0, stream >> > (Y);
		hipStreamSynchronize(stream);
		printf("\n\n");
	}
	*/

	// X = X * Y
	ButOnGPU << <1, limit, 0, stream >> > (X, d_rev);
	ButOnGPU << <1, limit, 0, stream >> > (Y, d_rev);
	NttOnGPU << <1, limit, 0, stream >> > (X, d_root, limit);
	NttOnGPU << <1, limit, 0, stream >> > (Y, d_root, limit);
	MulOnGPU << <1, limit, 0, stream >> > (X, Y);
	ButOnGPU << <1, limit, 0, stream >> > (X, d_rev);
	_NttOnGPU << <1, limit, 0, stream >> > (X, d_root, limit);
	MulDivOnGPU << <1, limit, 0, stream >> > (X, inv);
	PolynominalCarry << <1, 1, 0, stream >> > (X, limit / 2);

	// Y = X * _n mod R
	PolynominalModR << <1, limit / 2, 0, stream >> > (X, Y, limit / 2, nbits);
	ButOnGPU << <1, limit, 0, stream >> > (Y, d_rev);
	NttOnGPU << <1, limit, 0, stream >> > (Y, d_root, limit);
	MulOnGPU << <1, limit, 0, stream >> > (Y, _n_ntt);
	ButOnGPU << <1, limit, 0, stream >> > (Y, d_rev);
	_NttOnGPU << <1, limit, 0, stream >> > (Y, d_root, limit);
	MulDivOnGPU << <1, limit, 0, stream >> > (Y, inv);
	PolynominalCarry << <1, 1, 0, stream >> > (Y, limit / 2);
	PolynominalModR << <1, limit / 2, 0, stream >> > (Y, Y, limit / 2, nbits);

	// Y = Y * n
	ButOnGPU << <1, limit, 0, stream >> > (Y, d_rev);
	NttOnGPU << <1, limit, 0, stream >> > (Y, d_root, limit);
	MulOnGPU << <1, limit, 0, stream >> > (Y, n_ntt);
	ButOnGPU << <1, limit, 0, stream >> > (Y, d_rev);
	_NttOnGPU << <1, limit, 0, stream >> > (Y, d_root, limit);
	MulDivOnGPU << <1, limit, 0, stream >> > (Y, inv);

	// out = (X + Y) / R 
	NttAddOnGPU << <1, limit, 0, stream >> > (X, Y);
	PolynominalCarry << <1, 1, 0, stream >> > (X, limit - 1);
	PolynomialToBignum << <1, limit / 2, 0, stream >> > ((uint8_t*)out, X, limit / 2, nbits >> 3, nbits & 0b111);

	// if out >= n ,return out - n,else return out
	CmpOnGPU << <1, 1, 0, stream >> > (out, M, (limit - 1) / 8, flag);
	SubOnGPU << < 1, (limit + 7) / 8, ((limit + 7) / 8 + 1) * sizeof(uint32_t), stream >> > (out, M, flag);
}

// montgomery red with ntt
void NTTMonRedOnGPU(hipStream_t stream, uint8_t* y, uint32_t* out, uint32_t* X, uint32_t* Y, int limit, int* flag)
{
	// X = Y * 1
	BignumToPolynomial << <1, limit / 2, 0, stream >> > (y, X, limit / 2);
	BignumToPolynomial << <1, limit / 2, 0, stream >> > (y, Y, limit / 2);

	// Y = Y * _n mod R
	ButOnGPU << <1, limit, 0, stream >> > (Y, d_rev);
	NttOnGPU << <1, limit, 0, stream >> > (Y, d_root, limit);
	MulOnGPU << <1, limit, 0, stream >> > (Y, _n_ntt);
	ButOnGPU << <1, limit, 0, stream >> > (Y, d_rev);
	_NttOnGPU << <1, limit, 0, stream >> > (Y, d_root, limit);
	MulDivOnGPU << <1, limit, 0, stream >> > (Y, inv);
	PolynominalCarry << <1, 1, 0, stream >> > (Y, limit / 2);
	PolynominalModR << <1, limit / 2, 0, stream >> > (Y, Y, limit / 2, nbits);

	// Y = Y * n
	ButOnGPU << <1, limit, 0, stream >> > (Y, d_rev);
	NttOnGPU << <1, limit, 0, stream >> > (Y, d_root, limit);
	MulOnGPU << <1, limit, 0, stream >> > (Y, n_ntt);
	ButOnGPU << <1, limit, 0, stream >> > (Y, d_rev);
	_NttOnGPU << <1, limit, 0, stream >> > (Y, d_root, limit);
	MulDivOnGPU << <1, limit, 0, stream >> > (Y, inv);

	// out = (X + Y) / R
	NttAddOnGPU << <1, limit, 0, stream >> > (X, Y);
	PolynominalCarry << <1, 1, 0, stream >> > (X, limit - 1);
	PolynomialToBignum << <1, limit / 2, 0, stream >> > ((uint8_t*)out, X, limit / 2, nbits >> 3, nbits & 0b111);

	// if out >= n ,return out - n,else return out
	CmpOnGPU << <1, 1, 0, stream >> > (out, M, (limit - 1) / 8, flag);
	SubOnGPU << < 1, (limit + 7) / 8, ((limit + 7) / 8 + 1) * sizeof(uint32_t), stream >> > (out, M, flag);
}

// montgomery squ with ntt
void NTTMonSquOnGPU(hipStream_t stream, uint8_t* x, uint32_t* out, uint32_t* X, uint32_t* Y, int limit, int* flag)
{
	BignumToPolynomial << <1, limit / 2, 0, stream >> > (x, X, limit / 2);

	// X = X * Y
	ButOnGPU << <1, limit, 0, stream >> > (X, d_rev);
	NttOnGPU << <1, limit, 0, stream >> > (X, d_root, limit);
	MulOnGPU << <1, limit, 0, stream >> > (X, X);
	ButOnGPU << <1, limit, 0, stream >> > (X, d_rev);
	_NttOnGPU << <1, limit, 0, stream >> > (X, d_root, limit);
	MulDivOnGPU << <1, limit, 0, stream >> > (X, inv);;
	PolynominalCarry << <1, 1, 0, stream >> > (X, limit / 2);

	// Y = X * _n mod R
	PolynominalModR << <1, limit / 2, 0, stream >> > (X, Y, limit / 2, nbits);
	ButOnGPU << <1, limit, 0, stream >> > (Y, d_rev);
	NttOnGPU << <1, limit, 0, stream >> > (Y, d_root, limit);
	MulOnGPU << <1, limit, 0, stream >> > (Y, _n_ntt);
	ButOnGPU << <1, limit, 0, stream >> > (Y, d_rev);
	_NttOnGPU << <1, limit, 0, stream >> > (Y, d_root, limit);
	MulDivOnGPU << <1, limit, 0, stream >> > (Y, inv);
	PolynominalCarry << <1, 1, 0, stream >> > (Y, limit / 2);
	PolynominalModR << <1, limit / 2, 0, stream >> > (Y, Y, limit / 2, nbits);

	// Y = Y * n
	ButOnGPU << <1, limit, 0, stream >> > (Y, d_rev);
	NttOnGPU << <1, limit, 0, stream >> > (Y, d_root, limit);
	MulOnGPU << <1, limit, 0, stream >> > (Y, n_ntt);
	ButOnGPU << <1, limit, 0, stream >> > (Y, d_rev);
	_NttOnGPU << <1, limit, 0, stream >> > (Y, d_root, limit);
	MulDivOnGPU << <1, limit, 0, stream >> > (Y, inv);

	// out = (X + Y) / R 
	NttAddOnGPU << <1, limit, 0, stream >> > (X, Y);
	PolynominalCarry << <1, 1, 0, stream >> > (X, limit - 1);
	PolynomialToBignum << <1, limit / 2, 0, stream >> > ((uint8_t*)out, X, limit / 2, nbits >> 3, nbits & 0b111);

	// if out >= n ,return out - n,else return out
	CmpOnGPU << <1, 1, 0, stream >> > (out, M, (limit - 1) / 8, flag);
	SubOnGPU << < 1, (limit + 7) / 8, ((limit + 7) / 8 + 1) * sizeof(uint32_t), stream >> > (out, M, flag);
}


//input x e limit:nttitems ebits s:stream idex
//output:x^e mod n
void MonExpOnGPU(uint8_t* x, uint32_t* e, int limit, int ebits, int s)
{
	// x=x*r mod n
	NTTMonMulOnGPU(stream[s], x, y, (uint32_t*)xx[s], XX[s], YY[s], limit, flag[s]);

	// y=r mod n
	NTTMonRedOnGPU(stream[s], y, (uint32_t*)yy[s], XX[s], YY[s], limit, flag[s]);

	for (int i = 0; i < ebits - 1; i++)
	{
		/*
		if (i < 0)
		{
			test = 0;
			PrintOnGpu << <1, limit / 4, 0, stream[s] >> > ((uint32_t*)xx[s]);
			hipStreamSynchronize(stream[s]);
			printf("\n\n");
		}
		*/
		if ((e[i >> 5] >> (i & 0b11111)) & 1)
			NTTMonMulOnGPU(stream[s], xx[s], yy[s], (uint32_t*)yy[s], XX[s], YY[s], limit, flag[s]);
		//NTTMonMulOnGPU(stream[s], xx[s], xx[s], (uint32_t*)xx[s], XX[s], YY[s], limit, flag[s], test);
		NTTMonSquOnGPU(stream[s], xx[s], (uint32_t*)xx[s], XX[s], YY[s], limit, flag[s]);
		
	}

	NTTMonMulOnGPU(stream[s], xx[s], yy[s], (uint32_t*)yy[s], XX[s], YY[s], limit, flag[s]);
	NTTMonRedOnGPU(stream[s], yy[s], Out[s], XX[s], YY[s], limit, flag[s]);
	
}

//Pre calculation bignum to ntt
void NumToNtt(uint8_t* x,uint32_t* X, int* d_rev, ll* d_root, int limit)
{
	BignumToPolynomial << <1, limit / 2 >> > (x, X, limit / 2);
	ButOnGPU << <1, limit>> > (X, d_rev);
	NttOnGPU << <1, limit>> > (X, d_root, limit);
}

int initongpu()
{
	CHECK(hipHostMalloc((uint32_t * *)& nn, 256));
	CHECK(hipHostMalloc((uint32_t * *)& e, 256));
	CHECK(hipHostMalloc((uint32_t * *)& d, 256));
	CHECK(hipHostMalloc((uint32_t * *)& r2m, 256));
	CHECK(hipHostMalloc((uint32_t * *)& _n, 256));
	CHECK(hipHostMalloc((uint32_t * *)& val, 256));
	
	
	CHECK(hipMalloc((uint32_t * *)& n_ntt, 2048));
	CHECK(hipMalloc((uint32_t * *)& _n_ntt, 2048));
	CHECK(hipMalloc((uint8_t * *)& x, 256));
	CHECK(hipMalloc((uint8_t * *)& y, 256));
	CHECK(hipMalloc((uint32_t * *)& M, 256));
	CHECK(hipMemset(x, 0, 256));
	CHECK(hipMemset(y, 0, 256));
	CHECK(hipMemset(M, 0, 256));
	

	for (int i = 0; i < 16; i++)
	{
		CHECK(hipMalloc((uint32_t * *)& XX[i], 2048));
		CHECK(hipMalloc((uint32_t * *)& YY[i], 2048));
		CHECK(hipMemset(XX[i], 0, 2048));
		CHECK(hipMemset(XX[i], 0, 2048));

		CHECK(hipMalloc((uint8_t * *)& xx[i], 256));
		CHECK(hipMalloc((uint8_t * *)& yy[i], 256));
		CHECK(hipMemset(xx[i], 0, 256));
		CHECK(hipMemset(yy[i], 0, 256));

		CHECK(hipMalloc((uint32_t * *)& Out[i], 256));
		CHECK(hipMemset(Out[i], 0, 256));

		hipStreamCreate(&stream[i]);
		CHECK(hipMalloc((int**)& flag[i], 4))
	}
	
	//R = 2^1023 nbits = 1024
	char str1[] = "79eec1e33a41bf4592557bb1991b1830d4b445f55e3c9e683afc7a7f4abf05549a5e7ea811f8c3faf58450c2eafce1a25c5eb49821d0f930247ef2c6a6e426f01f91a6090292a433d84b93a1e6c5ba933c48f48923aa727f3de18c5fa4f1c0f7cce43cf407f94ee1d316d572b4428c7399158b76fa15f8b3dfbb36bd5f4bc5d1";
	bignum_from_string(nn, str1, 256);
	char str2[] = "233c05371e4c85731b382c88438ffacb918b8e73bb099554d546c43728684ea805fbac69f0d78bfa671c17225c393b1269d2cc28f20cab1568566edd4cb8bd2f59e4b25f4b3787af54e002216bc42a34a2bdbd7bfe4ddab35dde5256fc7bfbc1b39f641c86e99950768214e69b18f806b0d200908484eb7cf6e817ab57400861";
	bignum_from_string(d, str2, 256);
	char str3[] = "4e29e645da6efddda068a8dcfceea970a5e86f7b518655cd3fba103d6899618a6b7caa86df16f28f7bdadbe2ad250794c9f20c9c42338624ab077f9f9ae3733a5c3bf8b4686b56cfe635be0010bf734fdc2a4f2ce5cf920fd4e79c6b7330a8fc2025e61d33dd8b3056390a2226d9d9eaec37f7aea1682f25120c260ecb165823";
	bignum_from_string(val, str3, 256);
	char r2ms[] = "1a32ca1d9343f9ac08567501d91b0b29540e5e6914aaf46c460b92007b6264ca7a4be15e5346933dd2865022a2535729ea817c215f80714384b8235705b88bc3a295fe00ae789bd241d5816e5d617c362a2ed1bdd8b45ca26f558a987de829afe0253c33b6a7bab59c35429c29c4ab63a0ab16c7f8c4b9319f6f1947266522a5";
	bignum_from_string(r2m, r2ms, 256);
	char str4[] = "3de12848fef2d6ddfaff968d0baf084bb5298ef79a9d6b5d8d36dc8d91d21778cd2b0258797ebe7662c5c5167d8ada581ff567183a54e2d8d9e63d51b8e95b66723a1345c434cba4e7f12ea1d5aa66eeb1d44be48b3779ba4b42537da959cc019a313c3950de5e809860a3f32e6214f445c56a8fb5c8ccb60d06b3d2b6314ccf";
	bignum_from_string(_n, str4, 256);
	e[0] = 65537;

	/*
	//R = 2^511 nbits = 512
	char str1[] = "758463d46999c11496449db8dddd1e407de2e9a8f33612f454866acddd759da8173d4e3fe8c4eaf121f86f87ac8e1d58f54e2c6a80bcf8c404884795252224ad";
	bignum_from_string(nn, str1, 128);
	char str2[] = "68827b718d1452d4e72a5085f6b14dd516df34e3ae9fb94d96da0fa3d33e651cc244b0275a24ab0753b5c01eac2f8f0d700c587bbd6d8aeb6a4e99e1a9372655";
	bignum_from_string(d, str2, 128);
	char str3[] = "45462476f31c3dfde5ac5fde4862d33d917f52255d80555b543584a32b71762a1fc719a341c0e925e9fff02a657764ae78b143d324cfc8892695c55801237885";
	bignum_from_string(val, str3, 128);
	char r2ms[] = "47395beb0ae85106f9f8548040a9b165d9a37499d0d98a14a5bcd0b943d0549be18b2ced65bfc42db40331f3ec67faf9cccf19e51d3ef7a09e03ebb1855d5e5e";
	bignum_from_string(r2m, r2ms, 128);
	char str4[] = "6629e97280ecccca1530ec12f59413fb353ccfb99a050e1203b3c9df6753928d7adfc28193416c314ff21d8c00e17814d09dccfa2850d057ceef739ce5a920db";
	bignum_from_string(_n, str4, 128);
	e[0] = 65537;
	*/
	//R = 2^12 nbits = 12
	//nn[0] = 3233;
	//e[0] = 17;
	//d[0] = 2753;
	//r2m[0] = 1179;
	//val[0] = 855;
	//_n[0] = 2207;

	ebits = bignum_numbits(e);
	dbits = bignum_numbits(d);
	nbits = bignum_numbits(nn);

	int limit = 1;
	int n = nbits / 8, m = nbits / 8, L = -1;
	for (; limit <= n + m; limit <<= 1, L++);
	CHECK(hipMalloc((int**)& d_rev, limit * sizeof(int)));
	CHECK(hipMalloc((ll * *)& d_root, limit * sizeof(ll)));

	for (int i = 0; i < limit; i++)
		rev[i] = (rev[i >> 1] >> 1) | ((i & 1) << L);
	CHECK(hipMemcpy(d_rev, &rev, limit * sizeof(int), hipMemcpyHostToDevice));

	inv = power(limit, P - 2);
	ll temp_w = power(G, (P - 1) / limit);
	root[0] = 1;
	for (int i = 1; i < limit; ++i)
	{
		root[i] = root[i - 1] * temp_w % P;
	}
	CHECK(hipMemcpy(d_root, root, limit * sizeof(ll), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(x, val, limit / 2 * sizeof(uint8_t), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(y, r2m, limit / 2 * sizeof(uint8_t), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(M, _n, limit / 2 * sizeof(uint8_t), hipMemcpyHostToDevice));
	NumToNtt((uint8_t*)M, _n_ntt, d_rev, d_root, limit);
	CHECK(hipMemcpy(M, nn, limit / 2 * sizeof(uint8_t), hipMemcpyHostToDevice));
	NumToNtt((uint8_t*)M, n_ntt, d_rev, d_root, limit);
	CHECK(hipDeviceSynchronize());
	return limit;
}

void freemem()
{
	hipHostFree(nn);
	hipHostFree(e);
	hipHostFree(d);
	hipHostFree(r2m);
	hipHostFree(_n);
	hipHostFree(val);

	hipFree(n_ntt);
	hipFree(_n_ntt);
	hipFree(x);
	hipFree(y);
	hipFree(M);

	for (int i = 0; i < 16; i++)
	{
		hipFree(XX[i]);
		hipFree(YY[i]);
		hipFree(xx[i]);
		hipFree(yy[i]);
		hipFree(Out[i]);
		hipFree(flag[i]);
		hipStreamDestroy(stream[i]);
	}

	hipFree(d_rev);
	hipFree(d_root);

}

int main()
{
	double iStart, iElaps;
	int limit = 1;
	iStart = seconds();
	limit = initongpu();
	iElaps = seconds() - iStart;
	cout << "Initial time elapsed" << iElaps << "sec" << endl;

	iStart = seconds();
	for (int i = 0; i < 16; i++)
	{
		MonExpOnGPU(x, d, limit, dbits, i);
	}
	iElaps = seconds() - iStart;
	cout << "GPU encrypt time elapsed" << iElaps << "sec" << endl;


	iStart = seconds();
	for (int i = 0; i < 16; i++)
	{
		hipStreamSynchronize(stream[i]);
		MonExpOnGPU((uint8_t*)Out[i], e, limit, ebits, i);
	}
	iElaps = seconds() - iStart;
	cout << "GPU decrypt time elapsed" << iElaps << "sec" << endl;

	for (int i = 0; i < 16; i++)
	{
		PrintOnGpu << <1, limit / 2, 0, stream[i] >> > (Out[i]);
		hipStreamSynchronize(stream[i]);
		printf("\n\n");
	}
	
	freemem();
}